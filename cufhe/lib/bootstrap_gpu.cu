#include "hip/hip_runtime.h"
/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <include/cufhe.h>
#include <stdio.h>
#include <include/bootstrap_gpu.cuh>
#include <include/details/error_gpu.cuh>
#include <include/ntt_gpu/ntt.cuh>

#include <iostream>
using namespace std;

namespace cufhe {

using BootstrappingKeyNTT = TGSWSampleArray_T<FFP>;
BootstrappingKeyNTT* bk_ntt = nullptr;
MemoryDeleter bk_ntt_deleter = nullptr;
KeySwitchingKey* ksk_dev = nullptr;
MemoryDeleter ksk_dev_deleter = nullptr;
CuNTTHandler<>* ntt_handler = nullptr;

__global__ void __BootstrappingKeyToNTT__(BootstrappingKeyNTT bk_ntt,
                                          BootstrappingKey bk,
                                          CuNTTHandler<> ntt)
{
    __shared__ FFP sh_temp[1024];

    TGSWSample tgsw;
    bk.ExtractTGSWSample(&tgsw, blockIdx.z);
    TLWESample tlwe;
    tgsw.ExtractTLWESample(&tlwe, blockIdx.y);
    Torus* poly_in = tlwe.ExtractPoly(blockIdx.x);

    TGSWSample_T<FFP> tgsw_ntt;
    bk_ntt.ExtractTGSWSample(&tgsw_ntt, blockIdx.z);
    TLWESample_T<FFP> tlwe_ntt;
    tgsw_ntt.ExtractTLWESample(&tlwe_ntt, blockIdx.y);
    FFP* poly_out = tlwe_ntt.ExtractPoly(blockIdx.x);
    ntt.NTT<Torus>(poly_out, poly_in, sh_temp, 0);
}

void BootstrappingKeyToNTT(const BootstrappingKey* bk)
{
    BootstrappingKey* d_bk;
    d_bk = new BootstrappingKey(bk->n(), bk->k(), bk->l(), bk->w(), bk->t());
    std::pair<void*, MemoryDeleter> pair;
    pair = AllocatorGPU::New(d_bk->SizeMalloc());
    d_bk->set_data((BootstrappingKey::PointerType)pair.first);
    MemoryDeleter d_bk_deleter = pair.second;
    CuSafeCall(hipMemcpy(d_bk->data(), bk->data(), d_bk->SizeMalloc(),
                          hipMemcpyHostToDevice));

    Assert(bk_ntt == nullptr);
    bk_ntt =
        new BootstrappingKeyNTT(bk->n(), bk->k(), bk->l(), bk->w(), bk->t());
    pair = AllocatorGPU::New(bk_ntt->SizeMalloc());
    bk_ntt->set_data((BootstrappingKeyNTT::PointerType)pair.first);
    bk_ntt_deleter = pair.second;

    Assert(ntt_handler == nullptr);
    ntt_handler = new CuNTTHandler<>();
    ntt_handler->Create();
    ntt_handler->CreateConstant();
    hipDeviceSynchronize();
    CuCheckError();

    dim3 grid(bk->k() + 1, (bk->k() + 1) * bk->l(), bk->t());
    dim3 block(128);
    __BootstrappingKeyToNTT__<<<grid, block>>>(*bk_ntt, *d_bk, *ntt_handler);
    hipDeviceSynchronize();
    CuCheckError();

    d_bk_deleter(d_bk->data());
    delete d_bk;
}

void DeleteBootstrappingKeyNTT()
{
    bk_ntt_deleter(bk_ntt->data());
    delete bk_ntt;
    bk_ntt = nullptr;

    ntt_handler->Destroy();
    delete ntt_handler;
}

void KeySwitchingKeyToDevice(const KeySwitchingKey* ksk)
{
    Assert(ksk_dev == nullptr);
    ksk_dev = new KeySwitchingKey(ksk->n(), ksk->l(), ksk->w(), ksk->m());
    std::pair<void*, MemoryDeleter> pair;
    pair = AllocatorGPU::New(ksk_dev->SizeMalloc());
    ksk_dev->set_data((KeySwitchingKey::PointerType)pair.first);
    ksk_dev_deleter = pair.second;
    CuSafeCall(hipMemcpy(ksk_dev->data(), ksk->data(), ksk->SizeMalloc(),
                          hipMemcpyHostToDevice));
}

void DeleteKeySwitchingKey()
{
    ksk_dev_deleter(ksk_dev->data());
    delete ksk_dev;
    ksk_dev = nullptr;
}

__device__ inline uint32_t ModSwitch2048(uint32_t a)
{
    return (((uint64_t)a << 32) + (0x1UL << 52)) >> 53;
}

template <uint32_t lwe_n = 500, uint32_t tlwe_n = 1024,
          uint32_t decomp_bits = 2, uint32_t decomp_size = 8>
__device__ inline void KeySwitch(Torus* lwe, Torus* tlwe, Torus* ksk)
{
    static const Torus decomp_mask = (1u << decomp_bits) - 1;
    static const Torus decomp_offset = 1u << (31 - decomp_size * decomp_bits);
    uint32_t tid = ThisThreadRankInBlock();
    uint32_t bdim = ThisBlockSize();
    Torus tmp;
    Torus res = 0;
    Torus val = 0;
#pragma unroll 0
    for (int i = tid; i <= lwe_n; i += bdim) {
        if (i == lwe_n) res = tlwe[tlwe_n];
#pragma unroll 0
        for (int j = 0; j < tlwe_n; j++) {
            if (j == 0)
                tmp = tlwe[0];
            else
                tmp = -tlwe[1024 - j];
            tmp += decomp_offset;
            for (int k = 0; k < decomp_size; k++) {
                val = (tmp >> (32 - (k + 1) * decomp_bits)) & decomp_mask;
                if (val != 0)
                    res -= ksk[(j << 14) | (k << 11) | (val << 9) | i];
            }
        }
        lwe[i] = res;
    }
}

template <uint32_t lwe_n = 500, uint32_t tlwe_n = 1024, uint32_t tlwe_nbit = 10>
__device__ inline void RotatedTestVector(Torus* tlwe, int32_t bar, uint32_t mu)
{
    register uint32_t tid = ThisThreadRankInBlock();
    register uint32_t bdim = ThisBlockSize();
    register uint32_t cmp, neg, pos;
#pragma unroll
    for (int i = tid; i < tlwe_n; i += bdim) {
        tlwe[i] = 0;  // part a
        if (bar == 2 * tlwe_n)
            tlwe[i + tlwe_n] = mu;
        else {
            cmp = (uint32_t)(i < (bar & 1023));
            neg = -(cmp ^ (bar >> tlwe_nbit));
            pos = -((1 - cmp) ^ (bar >> tlwe_nbit));
            tlwe[i + tlwe_n] = (mu & pos) + ((-mu) & neg);  // part b
        }
    }
    __syncthreads();
}

__device__ void Accumulate(Torus* tlwe, FFP* sh_acc_ntt, FFP* sh_res_ntt,
                           uint32_t a_bar, FFP* tgsw_ntt, CuNTTHandler<> ntt)
{
    static const uint32_t decomp_bits = DEF_Bgbit;
    static const uint32_t decomp_mask = (1 << decomp_bits) - 1;
    static const int32_t decomp_half = 1 << (decomp_bits - 1);
    static const uint32_t decomp_offset =
        (0x1u << 31) + (0x1u << (31 - decomp_bits));
    uint32_t tid = ThisThreadRankInBlock();
    uint32_t bdim = ThisBlockSize();

    // temp[2] = sh_acc[2] * (x^exp - 1)
    // sh_acc_ntt[0, 1] = Decomp(temp[0])
    // sh_acc_ntt[2, 3] = Decomp(temp[1])
    // This algorithm is tested in cpp.
    Torus temp;
#pragma unroll
    for (int i = tid; i < 1024; i += bdim) {
        uint32_t cmp = (uint32_t)(i < (a_bar & 1023));
        uint32_t neg = -(cmp ^ (a_bar >> 10));
        uint32_t pos = -((1 - cmp) ^ (a_bar >> 10));
#pragma unroll
        for (int j = 0; j < 2; j++) {
            temp = tlwe[(j << 10) | ((i - a_bar) & 1023)];
            temp = (temp & pos) + ((-temp) & neg);
            temp -= tlwe[(j << 10) | i];
            // decomp temp
            temp += decomp_offset;
            sh_acc_ntt[(2 * j) * 1024 + i] = FFP(Torus(
                ((temp >> (32 - decomp_bits)) & decomp_mask) - decomp_half));
            sh_acc_ntt[(2 * j + 1) * 1024 + i] =
                FFP(Torus(((temp >> (32 - 2 * decomp_bits)) & decomp_mask) -
                          decomp_half));
        }
    }
    __syncthreads();  // must

    // 4 NTTs with 512 threads.
    // Input/output/buffer use the same shared memory location.
    if (tid < 512) {
        FFP* tar = &sh_acc_ntt[tid >> 7 << 10];
        ntt.NTT<FFP>(tar, tar, tar, tid >> 7 << 7);
    }
    else {  // must meet 4 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();

// Multiply with bootstrapping key in global memory.
#pragma unroll
    for (int i = tid; i < 1024; i += bdim) {
        sh_res_ntt[4096 + i] = 0;
#pragma unroll
        for (int j = 0; j < 4; j++)
            sh_res_ntt[4096 + i] +=
                sh_acc_ntt[j * 1024 + i] * tgsw_ntt[((2 * j + 1) << 10) + i];
    }
    __syncthreads();  // new
#pragma unroll
    for (int i = tid; i < 1024; i += bdim) {
        FFP temp = 0;
#pragma unroll
        for (int j = 0; j < 4; j++)
            temp += sh_acc_ntt[j * 1024 + i] * tgsw_ntt[((2 * j) << 10) + i];
        sh_res_ntt[i] = temp;
    }
    __syncthreads();  // must

    // 2 NTTInvs and add acc with 256 threads.
    if (tid < 256) {
        FFP* src = &sh_res_ntt[tid >> 7 << 12];
        ntt.NTTInvAdd<Torus>(&tlwe[tid >> 7 << 10], src, src, tid >> 7 << 7);
    }
    else {  // must meet 4 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();  // must
}

__global__ void __Bootstrap__(Torus* out, Torus* in, Torus mu, FFP* bk,
                              Torus* ksk, CuNTTHandler<> ntt)
{
    //  Assert(bk.k() == 1);
    //  Assert(bk.l() == 2);
    //  Assert(bk.n() == 1024);
    __shared__ FFP sh[6 * 1024];
    //  FFP* sh_acc_ntt[4] = { sh, sh + 1024, sh + 2048, sh + 3072 };
    //  FFP* sh_res_ntt[2] = { sh, sh + 4096 };
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector
    // acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar = 2048 - ModSwitch2048(in[500]);
    register uint32_t tid = ThisThreadRankInBlock();
    register uint32_t bdim = ThisBlockSize();
    register uint32_t cmp, neg, pos;
#pragma unroll
    for (int i = tid; i < 1024; i += bdim) {
        tlwe[i] = 0;  // part a
        if (bar == 2048)
            tlwe[i + 1024] = mu;
        else {
            cmp = (uint32_t)(i < (bar & 1023));
            neg = -(cmp ^ (bar >> 10));
            pos = -((1 - cmp) ^ (bar >> 10));
            tlwe[i + 1024] = (mu & pos) + ((-mu) & neg);  // part b
        }
    }
    __syncthreads();
// accumulate
#pragma unroll
    for (int i = 0; i < 500; i++) {  // 500 iterations
        bar = ModSwitch2048(in[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }

    static const uint32_t lwe_n = 500;
    static const uint32_t tlwe_n = 1024;
    static const uint32_t ks_bits = 2;
    static const uint32_t ks_size = 8;
    KeySwitch<lwe_n, tlwe_n, ks_bits, ks_size>(out, tlwe, ksk);
}

__global__ void __NandBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                  Torus fix, FFP* bk, Torus* ksk,
                                  CuNTTHandler<> ntt)
{
    __shared__ FFP sh[(2 * DEF_l + 2) * DEF_N];  // This is V100's MAX
    // Use Last section to hold tlwe. This may to make these data in serial
    Torus* tlwe = (Torus*)&sh[(2 * DEF_l + 1) * DEF_N];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix - in0[DEF_n] - in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - in0[i] - in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __OrBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                Torus fix, FFP* bk, Torus* ksk,
                                CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix + in0[DEF_n] + in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < 500; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + in0[i] + in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __OrYNBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                  Torus fix, FFP* bk, Torus* ksk,
                                  CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix + in0[DEF_n] - in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + in0[i] - in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __OrNYBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                  Torus fix, FFP* bk, Torus* ksk,
                                  CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix - in0[DEF_n] + in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - in0[i] + in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __AndBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                 Torus fix, FFP* bk, Torus* ksk,
                                 CuNTTHandler<> ntt)
{
    __shared__ FFP sh[(2 * DEF_l + 2) * DEF_N];  // This is V100's MAX
    Torus* tlwe = (Torus*)&sh[(2 * DEF_l + 1) * DEF_N];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix + in0[DEF_n] + in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + in0[i] + in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __AndYNBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                   Torus fix, FFP* bk, Torus* ksk,
                                   CuNTTHandler<> ntt)
{
    __shared__ FFP sh[(2 * DEF_l + 2) * DEF_N];  // This is V100's MAX
    Torus* tlwe = (Torus*)&sh[(2 * DEF_l + 1) * DEF_N];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix + in0[DEF_n] - in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + in0[i] - in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __AndNYBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                   Torus fix, FFP* bk, Torus* ksk,
                                   CuNTTHandler<> ntt)
{
    __shared__ FFP sh[(2 * DEF_l + 2) * DEF_N];  // This is V100's MAX
    Torus* tlwe = (Torus*)&sh[(2 * DEF_l + 1) * DEF_N];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix - in0[DEF_n] + in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - in0[i] + in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __NorBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                 Torus fix, FFP* bk, Torus* ksk,
                                 CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar = 2048 - ModSwitch2048(fix - in0[500] - in1[500]);
    RotatedTestVector<DEF_n, DEF_N>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < 500; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - in0[i] - in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __XorBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                 Torus fix, FFP* bk, Torus* ksk,
                                 CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2048 - ModSwitch2048(fix + 2 * in0[500] + 2 * in1[500]);
    RotatedTestVector<DEF_n, DEF_N>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < 500; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + 2 * in0[i] + 2 * in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __XnorBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                  Torus fix, FFP* bk, Torus* ksk,
                                  CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2048 - ModSwitch2048(fix - 2 * in0[500] - 2 * in1[500]);
    RotatedTestVector<DEF_n, DEF_N>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - 2 * in0[i] - 2 * in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __NotBootstrap__(Torus* out, Torus* in, int n)
{
#pragma unroll
    for (int i = 0; i <= n; i++) {
        out[i] = -in[i];
    }
    __syncthreads();
}

// Mux(inc,in1,in0) = inc?in1:in0 = inc&in1 + (!inc)&in0
__global__ void __MuxBootstrap__(Torus* out, Torus* inc, Torus* in1, Torus* in0,
                                 Torus mu, Torus fix, Torus muxfix, FFP* bk,
                                 Torus* ksk, CuNTTHandler<> ntt)
{
    __shared__ FFP sh[(2 * DEF_l + 2) * DEF_N];  // This is V100's MAX
    // Use Last section to hold tlwe. This may to make these data in serial
    Torus* tlwe = (Torus*)&sh[(2 * DEF_l + 1) * DEF_N];
    Torus temp[DEF_N + 1];
    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix + inc[DEF_n] + in1[DEF_n]);
    register uint32_t tid = ThisThreadRankInBlock();
    register uint32_t bdim = ThisBlockSize();
    RotatedTestVector<DEF_n, DEF_N>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + inc[i] + in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }

#pragma unroll
    for (int i = tid; i <= DEF_N; i += bdim) {
        temp[i] = tlwe[i];
    }

    __syncthreads();

    bar = 2 * DEF_N - ModSwitch2048(fix - inc[DEF_n] + in0[DEF_n]);

    RotatedTestVector<DEF_n, DEF_N>(tlwe, bar, mu);

#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - inc[i] + in0[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }

#pragma unroll
    for (int i = tid; i <= DEF_N; i += bdim) {
        tlwe[i] += temp[i];
        if (i == DEF_N) {
            tlwe[DEF_N] += muxfix;
        }
    }

    __syncthreads();

    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __NoiselessTrivial__(Torus* out, Torus pmu)
{
    register uint32_t tid = ThisThreadRankInBlock();
    register uint32_t bdim = ThisBlockSize();
#pragma unroll
    for (int i = tid; i <= DEF_n; i += bdim) {
        if (i == DEF_n)
            out[DEF_n] = pmu;
        else
            out[i] = 0;
    }
}

void Bootstrap(LWESample* out, LWESample* in, Torus mu, hipStream_t st)
{
    dim3 grid(1);
    dim3 block(512);
    __Bootstrap__<<<grid, block, 0, st>>>(out->data(), in->data(), mu,
                                          bk_ntt->data(), ksk_dev->data(),
                                          *ntt_handler);
    CuCheckError();
}

void NandBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                   Torus fix, hipStream_t st)
{
    __NandBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void OrBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                 Torus fix, hipStream_t st)
{
    __OrBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void OrYNBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                   Torus fix, hipStream_t st)
{
    __OrYNBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void OrNYBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                   Torus fix, hipStream_t st)
{
    __OrNYBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void AndBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                  Torus fix, hipStream_t st)
{
    __AndBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void AndYNBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                    Torus fix, hipStream_t st)
{
    __AndYNBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void AndNYBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                    Torus fix, hipStream_t st)
{
    __AndNYBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void NorBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                  Torus fix, hipStream_t st)
{
    __NorBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void XorBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                  Torus fix, hipStream_t st)
{
    __XorBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void XnorBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                   Torus fix, hipStream_t st)
{
    __XnorBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void NotBootstrap(LWESample* out, LWESample* in, int n, hipStream_t st)
{
    __NotBootstrap__<<<1, DEF_N / 2, 0, st>>>(out->data(), in->data(), n);
    CuCheckError();
}

void MuxBootstrap(LWESample* out, LWESample* inc, LWESample* in1,
                  LWESample* in0, Torus mu, Torus fix, Torus muxfix,
                  hipStream_t st)
{
    __MuxBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), inc->data(), in1->data(), in0->data(), mu, fix, muxfix,
        bk_ntt->data(), ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void NoiselessTrivial(LWESample* out, int p, Torus mu, hipStream_t st)
{
    __NoiselessTrivial__<<<1, DEF_n + 1, 0, st>>>(out->data(), p ? mu : -mu);
}
}  // namespace cufhe