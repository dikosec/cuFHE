#include "hip/hip_runtime.h"
/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <include/cufhe.h>
#include <stdio.h>
#include <include/bootstrap_gpu.cuh>
#include <include/details/error_gpu.cuh>
#include <include/ntt_gpu/ntt.cuh>

#include <iostream>
using namespace std;

namespace cufhe {

using BootstrappingKeyNTT = TGSWSampleArray_T<FFP>;
BootstrappingKeyNTT* bk_ntt = nullptr;
MemoryDeleter bk_ntt_deleter = nullptr;
KeySwitchingKey* ksk_dev = nullptr;
MemoryDeleter ksk_dev_deleter = nullptr;
CuNTTHandler<>* ntt_handler = nullptr;

__global__ void __BootstrappingKeyToNTT__(BootstrappingKeyNTT bk_ntt,
                                          BootstrappingKey bk,
                                          CuNTTHandler<> ntt)
{
    __shared__ FFP sh_temp[1024];

    TGSWSample tgsw;
    bk.ExtractTGSWSample(&tgsw, blockIdx.z);
    TLWESample tlwe;
    tgsw.ExtractTLWESample(&tlwe, blockIdx.y);
    Torus* poly_in = tlwe.ExtractPoly(blockIdx.x);

    TGSWSample_T<FFP> tgsw_ntt;
    bk_ntt.ExtractTGSWSample(&tgsw_ntt, blockIdx.z);
    TLWESample_T<FFP> tlwe_ntt;
    tgsw_ntt.ExtractTLWESample(&tlwe_ntt, blockIdx.y);
    FFP* poly_out = tlwe_ntt.ExtractPoly(blockIdx.x);
    ntt.NTT<Torus>(poly_out, poly_in, sh_temp, 0);
}

void BootstrappingKeyToNTT(const BootstrappingKey* bk)
{
    BootstrappingKey* d_bk;
    d_bk = new BootstrappingKey(bk->n(), bk->k(), bk->l(), bk->w(), bk->t());
    std::pair<void*, MemoryDeleter> pair;
    pair = AllocatorGPU::New(d_bk->SizeMalloc());
    d_bk->set_data((BootstrappingKey::PointerType)pair.first);
    MemoryDeleter d_bk_deleter = pair.second;
    CuSafeCall(hipMemcpy(d_bk->data(), bk->data(), d_bk->SizeMalloc(),
                          hipMemcpyHostToDevice));

    Assert(bk_ntt == nullptr);
    bk_ntt =
        new BootstrappingKeyNTT(bk->n(), bk->k(), bk->l(), bk->w(), bk->t());
    pair = AllocatorGPU::New(bk_ntt->SizeMalloc());
    bk_ntt->set_data((BootstrappingKeyNTT::PointerType)pair.first);
    bk_ntt_deleter = pair.second;

    Assert(ntt_handler == nullptr);
    ntt_handler = new CuNTTHandler<>();
    ntt_handler->Create();
    ntt_handler->CreateConstant();
    hipDeviceSynchronize();
    CuCheckError();

    dim3 grid(bk->k() + 1, (bk->k() + 1) * bk->l(), bk->t());
    dim3 block(128);
    __BootstrappingKeyToNTT__<<<grid, block>>>(*bk_ntt, *d_bk, *ntt_handler);
    hipDeviceSynchronize();
    CuCheckError();

    d_bk_deleter(d_bk->data());
    delete d_bk;
}

void DeleteBootstrappingKeyNTT()
{
    bk_ntt_deleter(bk_ntt->data());
    delete bk_ntt;
    bk_ntt = nullptr;

    ntt_handler->Destroy();
    delete ntt_handler;
}

void KeySwitchingKeyToDevice(const KeySwitchingKey* ksk)
{
    Assert(ksk_dev == nullptr);
    ksk_dev = new KeySwitchingKey(ksk->n(), ksk->l(), ksk->w(), ksk->m());
    std::pair<void*, MemoryDeleter> pair;
    pair = AllocatorGPU::New(ksk_dev->SizeMalloc());
    ksk_dev->set_data((KeySwitchingKey::PointerType)pair.first);
    ksk_dev_deleter = pair.second;
    CuSafeCall(hipMemcpy(ksk_dev->data(), ksk->data(), ksk->SizeMalloc(),
                          hipMemcpyHostToDevice));
}

void DeleteKeySwitchingKey()
{
    ksk_dev_deleter(ksk_dev->data());
    delete ksk_dev;
    ksk_dev = nullptr;
}

__device__ inline uint32_t ModSwitch2048(uint32_t a)
{
    return (((uint64_t)a << 32) + (0x1UL << 52)) >> 53;
}

template <uint32_t lwe_n = 500, uint32_t tlwe_n = 1024,
          uint32_t decomp_bits = 2, uint32_t decomp_size = 8>
__device__ inline void KeySwitch(Torus* lwe, Torus* tlwe, Torus* ksk)
{
    static const Torus decomp_mask = (1u << decomp_bits) - 1;
    static const Torus decomp_offset = 1u << (31 - decomp_size * decomp_bits);
    uint32_t tid = ThisThreadRankInBlock();
    uint32_t bdim = ThisBlockSize();
    Torus tmp;
    Torus res = 0;
    Torus val = 0;
#pragma unroll 0
    for (int i = tid; i <= lwe_n; i += bdim) {
        if (i == lwe_n) res = tlwe[tlwe_n];
#pragma unroll 0
        for (int j = 0; j < tlwe_n; j++) {
            if (j == 0)
                tmp = tlwe[0];
            else
                tmp = -tlwe[1024 - j];
            tmp += decomp_offset;
            for (int k = 0; k < decomp_size; k++) {
                val = (tmp >> (32 - (k + 1) * decomp_bits)) & decomp_mask;
                if (val != 0)
                    res -= ksk[(j << 14) | (k << 11) | (val << 9) | i];
            }
        }
        lwe[i] = res;
    }
}

template <uint32_t lwe_n = 500, uint32_t tlwe_n = 1024, uint32_t tlwe_nbit = 10>
__device__ inline void RotatedTestVector(Torus* tlwe, int32_t bar, uint32_t mu)
{
    // volatile is needed to make register usage of Mux to 128.
    // Reference https://devtalk.nvidia.com/default/topic/466758/cuda-programming-and-performance/tricks-to-fight-register-pressure-or-how-i-got-down-from-29-to-15-registers-/
    volatile uint32_t tid = ThisThreadRankInBlock();
    volatile uint32_t bdim = ThisBlockSize();
    uint32_t cmp, neg, pos;
#pragma unroll
    for (int i = tid; i < tlwe_n; i += bdim) {
        tlwe[i] = 0;  // part a
        if (bar == 2 * tlwe_n)
            tlwe[i + tlwe_n] = mu;
        else {
            cmp = (uint32_t)(i < (bar & 1023));
            neg = -(cmp ^ (bar >> tlwe_nbit));
            pos = -((1 - cmp) ^ (bar >> tlwe_nbit));
            tlwe[i + tlwe_n] = (mu & pos) + ((-mu) & neg);  // part b
        }
    }
    __syncthreads();
}

__device__ void Accumulate(Torus* tlwe, FFP* sh_acc_ntt, FFP* sh_res_ntt,
                           uint32_t a_bar, FFP* tgsw_ntt, CuNTTHandler<> ntt)
{
    static const uint32_t decomp_bits = DEF_Bgbit;
    static const uint32_t decomp_mask = (1 << decomp_bits) - 1;
    static const int32_t decomp_half = 1 << (decomp_bits - 1);
    static const uint32_t decomp_offset =
        (0x1u << 31) + (0x1u << (31 - decomp_bits));
    uint32_t tid = ThisThreadRankInBlock();
    uint32_t bdim = ThisBlockSize();

    // temp[2] = sh_acc[2] * (x^exp - 1)
    // sh_acc_ntt[0, 1] = Decomp(temp[0])
    // sh_acc_ntt[2, 3] = Decomp(temp[1])
    // This algorithm is tested in cpp.
    Torus temp;
#pragma unroll
    for (int i = tid; i < 1024; i += bdim) {
        uint32_t cmp = (uint32_t)(i < (a_bar & 1023));
        uint32_t neg = -(cmp ^ (a_bar >> 10));
        uint32_t pos = -((1 - cmp) ^ (a_bar >> 10));
#pragma unroll
        for (int j = 0; j < 2; j++) {
            temp = tlwe[(j << 10) | ((i - a_bar) & 1023)];
            temp = (temp & pos) + ((-temp) & neg);
            temp -= tlwe[(j << 10) | i];
            // decomp temp
            temp += decomp_offset;
            sh_acc_ntt[(2 * j) * 1024 + i] = FFP(Torus(
                ((temp >> (32 - decomp_bits)) & decomp_mask) - decomp_half));
            sh_acc_ntt[(2 * j + 1) * 1024 + i] =
                FFP(Torus(((temp >> (32 - 2 * decomp_bits)) & decomp_mask) -
                          decomp_half));
        }
    }
    __syncthreads();  // must

    // 4 NTTs with 512 threads.
    // Input/output/buffer use the same shared memory location.
    if (tid < 512) {
        FFP* tar = &sh_acc_ntt[tid >> 7 << 10];
        ntt.NTT<FFP>(tar, tar, tar, tid >> 7 << 7);
    }
    else {  // must meet 4 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();

// Multiply with bootstrapping key in global memory.
#pragma unroll
    for (int i = tid; i < 1024; i += bdim) {
        sh_res_ntt[4096 + i] = 0;
#pragma unroll
        for (int j = 0; j < 4; j++)
            sh_res_ntt[4096 + i] +=
                sh_acc_ntt[j * 1024 + i] * tgsw_ntt[((2 * j + 1) << 10) + i];
    }
    __syncthreads();  // new
#pragma unroll
    for (int i = tid; i < 1024; i += bdim) {
        FFP temp = 0;
#pragma unroll
        for (int j = 0; j < 4; j++)
            temp += sh_acc_ntt[j * 1024 + i] * tgsw_ntt[((2 * j) << 10) + i];
        sh_res_ntt[i] = temp;
    }
    __syncthreads();  // must

    // 2 NTTInvs and add acc with 256 threads.
    if (tid < 256) {
        FFP* src = &sh_res_ntt[tid >> 7 << 12];
        ntt.NTTInvAdd<Torus>(&tlwe[tid >> 7 << 10], src, src, tid >> 7 << 7);
    }
    else {  // must meet 4 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();  // must
}

__global__ void __Bootstrap__(Torus* out, Torus* in, Torus mu, FFP* bk,
                              Torus* ksk, CuNTTHandler<> ntt)
{
    //  Assert(bk.k() == 1);
    //  Assert(bk.l() == 2);
    //  Assert(bk.n() == 1024);
    __shared__ FFP sh[6 * 1024];
    //  FFP* sh_acc_ntt[4] = { sh, sh + 1024, sh + 2048, sh + 3072 };
    //  FFP* sh_res_ntt[2] = { sh, sh + 4096 };
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector
    // acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar = 2048 - ModSwitch2048(in[500]);
    register uint32_t tid = ThisThreadRankInBlock();
    register uint32_t bdim = ThisBlockSize();
    register uint32_t cmp, neg, pos;
#pragma unroll
    for (int i = tid; i < 1024; i += bdim) {
        tlwe[i] = 0;  // part a
        if (bar == 2048)
            tlwe[i + 1024] = mu;
        else {
            cmp = (uint32_t)(i < (bar & 1023));
            neg = -(cmp ^ (bar >> 10));
            pos = -((1 - cmp) ^ (bar >> 10));
            tlwe[i + 1024] = (mu & pos) + ((-mu) & neg);  // part b
        }
    }
    __syncthreads();
// accumulate
#pragma unroll
    for (int i = 0; i < 500; i++) {  // 500 iterations
        bar = ModSwitch2048(in[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }

    static const uint32_t lwe_n = 500;
    static const uint32_t tlwe_n = 1024;
    static const uint32_t ks_bits = 2;
    static const uint32_t ks_size = 8;
    KeySwitch<lwe_n, tlwe_n, ks_bits, ks_size>(out, tlwe, ksk);
}

__global__ void __NandBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                  Torus fix, FFP* bk, Torus* ksk,
                                  CuNTTHandler<> ntt)
{
    __shared__ FFP sh[(2 * DEF_l + 2) * DEF_N];  // This is V100's MAX
    // Use Last section to hold tlwe. This may to make these data in serial
    Torus* tlwe = (Torus*)&sh[(2 * DEF_l + 1) * DEF_N];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix - in0[DEF_n] - in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - in0[i] - in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __OrBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                Torus fix, FFP* bk, Torus* ksk,
                                CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix + in0[DEF_n] + in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < 500; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + in0[i] + in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __OrYNBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                  Torus fix, FFP* bk, Torus* ksk,
                                  CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix + in0[DEF_n] - in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + in0[i] - in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __OrNYBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                  Torus fix, FFP* bk, Torus* ksk,
                                  CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix - in0[DEF_n] + in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - in0[i] + in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __AndBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                 Torus fix, FFP* bk, Torus* ksk,
                                 CuNTTHandler<> ntt)
{
    __shared__ FFP sh[(2 * DEF_l + 2) * DEF_N];  // This is V100's MAX
    Torus* tlwe = (Torus*)&sh[(2 * DEF_l + 1) * DEF_N];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix + in0[DEF_n] + in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + in0[i] + in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __AndYNBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                   Torus fix, FFP* bk, Torus* ksk,
                                   CuNTTHandler<> ntt)
{
    __shared__ FFP sh[(2 * DEF_l + 2) * DEF_N];  // This is V100's MAX
    Torus* tlwe = (Torus*)&sh[(2 * DEF_l + 1) * DEF_N];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix + in0[DEF_n] - in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + in0[i] - in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __AndNYBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                   Torus fix, FFP* bk, Torus* ksk,
                                   CuNTTHandler<> ntt)
{
    __shared__ FFP sh[(2 * DEF_l + 2) * DEF_N];  // This is V100's MAX
    Torus* tlwe = (Torus*)&sh[(2 * DEF_l + 1) * DEF_N];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix - in0[DEF_n] + in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N, DEF_Nbit>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - in0[i] + in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __NorBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                 Torus fix, FFP* bk, Torus* ksk,
                                 CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar = 2048 - ModSwitch2048(fix - in0[500] - in1[500]);
    RotatedTestVector<DEF_n, DEF_N>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < 500; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - in0[i] - in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __XorBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                 Torus fix, FFP* bk, Torus* ksk,
                                 CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2048 - ModSwitch2048(fix + 2 * in0[500] + 2 * in1[500]);
    RotatedTestVector<DEF_n, DEF_N>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < 500; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + 2 * in0[i] + 2 * in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __XnorBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu,
                                  Torus fix, FFP* bk, Torus* ksk,
                                  CuNTTHandler<> ntt)
{
    __shared__ FFP sh[6 * 1024];
    Torus* tlwe = (Torus*)&sh[5120];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2048 - ModSwitch2048(fix - 2 * in0[500] - 2 * in1[500]);
    RotatedTestVector<DEF_n, DEF_N>(tlwe, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - 2 * in0[i] - 2 * in1[i]);
        Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
    }
    KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__ void __CopyBootstrap__(Torus* out, Torus* in, int n)
{
#pragma unroll
    for (int i = 0; i <= n; i++) {
        out[i] = in[i];
    }
    __syncthreads();
}

__global__ void __NotBootstrap__(Torus* out, Torus* in, int n)
{
#pragma unroll
    for (int i = 0; i <= n; i++) {
        out[i] = -in[i];
    }
    __syncthreads();
}

// Mux(inc,in1,in0) = inc?in1:in0 = inc&in1 + (!inc)&in0
__global__ void __MuxBootstrap__(Torus* out, Torus* inc, Torus* in1, Torus* in0,
                                 Torus mu, Torus fix, Torus muxfix, FFP* bk,
                                 Torus* ksk, CuNTTHandler<> ntt)
{
    // To use over 48 KiB shared Memory, the dynamic allocation is required.
    extern __shared__ FFP sh[];
    // Use Last section to hold tlwe. This may to make these data in serial
    Torus* tlwe1 = (Torus*)&sh[(2 * DEF_l + 1) * DEF_N];
    Torus* tlwe0 = (Torus*)&sh[(2 * DEF_l + 2) * DEF_N];
    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register int32_t bar =
        2 * DEF_N - ModSwitch2048(fix + inc[DEF_n] + in1[DEF_n]);
    RotatedTestVector<DEF_n, DEF_N>(tlwe1, bar, mu);

// accumulate
#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 + inc[i] + in1[i]);
        Accumulate(tlwe1, sh, sh, bar, bk + (i << 13), ntt);
    }

    bar = 2 * DEF_N - ModSwitch2048(fix - inc[DEF_n] + in0[DEF_n]);

    RotatedTestVector<DEF_n, DEF_N>(tlwe0, bar, mu);

#pragma unroll
    for (int i = 0; i < DEF_n; i++) {  // 500 iterations
        bar = ModSwitch2048(0 - inc[i] + in0[i]);
        Accumulate(tlwe0, sh, sh, bar, bk + (i << 13), ntt);
    }

    volatile uint32_t tid = ThisThreadRankInBlock();
    volatile uint32_t bdim = ThisBlockSize();
#pragma unroll
    for (int i = tid; i <= DEF_N; i += bdim) {
        tlwe1[i] += tlwe0[i];
        if (i == DEF_N) {
            tlwe1[DEF_N] += muxfix;
        }
    }

    __syncthreads();

    KeySwitch<500, 1024, 2, 8>(out, tlwe1, ksk);
}

__global__ void __NoiselessTrivial__(Torus* out, Torus pmu)
{
    register uint32_t tid = ThisThreadRankInBlock();
    register uint32_t bdim = ThisBlockSize();
#pragma unroll
    for (int i = tid; i <= DEF_n; i += bdim) {
        if (i == DEF_n)
            out[DEF_n] = pmu;
        else
            out[i] = 0;
    }
}

void Bootstrap(LWESample* out, LWESample* in, Torus mu, hipStream_t st)
{
    dim3 grid(1);
    dim3 block(512);
    __Bootstrap__<<<grid, block, 0, st>>>(out->data(), in->data(), mu,
                                          bk_ntt->data(), ksk_dev->data(),
                                          *ntt_handler);
    CuCheckError();
}

void NandBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                   Torus fix, hipStream_t st)
{
    __NandBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void OrBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                 Torus fix, hipStream_t st)
{
    __OrBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void OrYNBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                   Torus fix, hipStream_t st)
{
    __OrYNBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void OrNYBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                   Torus fix, hipStream_t st)
{
    __OrNYBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void AndBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                  Torus fix, hipStream_t st)
{
    __AndBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void AndYNBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                    Torus fix, hipStream_t st)
{
    __AndYNBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void AndNYBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                    Torus fix, hipStream_t st)
{
    __AndNYBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void NorBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                  Torus fix, hipStream_t st)
{
    __NorBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void XorBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                  Torus fix, hipStream_t st)
{
    __XorBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void XnorBootstrap(LWESample* out, LWESample* in0, LWESample* in1, Torus mu,
                   Torus fix, hipStream_t st)
{
    __XnorBootstrap__<<<1, DEF_N / 2, 0, st>>>(
        out->data(), in0->data(), in1->data(), mu, fix, bk_ntt->data(),
        ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void CopyBootstrap(LWESample* out, LWESample* in, int n, hipStream_t st)
{
    __CopyBootstrap__<<<1, DEF_N / 2, 0, st>>>(out->data(), in->data(), n);
    CuCheckError();
}

void NotBootstrap(LWESample* out, LWESample* in, int n, hipStream_t st)
{
    __NotBootstrap__<<<1, DEF_N / 2, 0, st>>>(out->data(), in->data(), n);
    CuCheckError();
}

void MuxBootstrap(LWESample* out, LWESample* inc, LWESample* in1,
                  LWESample* in0, Torus mu, Torus fix, Torus muxfix,
                  hipStream_t st)
{
    int maxbytes = 98304; // 96 KB
    hipFuncSetAttribute(reinterpret_cast<const void*>(__MuxBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize, (2*DEF_l + 3)*DEF_N*sizeof(FFP));
    __MuxBootstrap__<<<1, DEF_N / 2, (2*DEF_l + 3)*DEF_N*sizeof(FFP), st>>>(
        out->data(), inc->data(), in1->data(), in0->data(), mu, fix, muxfix,
        bk_ntt->data(), ksk_dev->data(), *ntt_handler);
    CuCheckError();
}

void NoiselessTrivial(LWESample* out, int p, Torus mu, hipStream_t st)
{
    __NoiselessTrivial__<<<1, DEF_n + 1, 0, st>>>(out->data(), p ? mu : -mu);
}
}  // namespace cufhe
