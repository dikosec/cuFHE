/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <include/cufhe.h>
#include <include/bootstrap_gpu.cuh>
#include <include/cufhe_gpu.cuh>

namespace cufhe {

void Initialize(const PubKey& pub_key)
{
    BootstrappingKeyToNTT(pub_key.bk_);
    KeySwitchingKeyToDevice(pub_key.ksk_);
}

void CleanUp()
{
    DeleteBootstrappingKeyNTT();
    DeleteKeySwitchingKey();
}

inline void CtxtCopyH2D(const Ctxt& c, Stream st)
{
    hipMemcpyAsync(c.lwe_sample_device_->data(), c.lwe_sample_->data(),
                    c.lwe_sample_->SizeData(), hipMemcpyHostToDevice, st.st());
}

inline void CtxtCopyD2H(const Ctxt& c, Stream st)
{
    hipMemcpyAsync(c.lwe_sample_->data(), c.lwe_sample_device_->data(),
                    c.lwe_sample_->SizeData(), hipMemcpyDeviceToHost, st.st());
}

void Nand(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(1, 8);
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    NandBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                  in1.lwe_sample_device_, mu, fix, st.st());
    CtxtCopyD2H(out, st);
}

void gNand(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(1, 8);
    NandBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                  in1.lwe_sample_device_, mu, fix, st.st());
}

void Or(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(1, 8);
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    OrBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                in1.lwe_sample_device_, mu, fix, st.st());
    CtxtCopyD2H(out, st);
}

void gOr(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(1, 8);
    OrBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                in1.lwe_sample_device_, mu, fix, st.st());
}

void OrYN(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(1, 8);
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    OrYNBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                  in1.lwe_sample_device_, mu, fix, st.st());
    CtxtCopyD2H(out, st);
}

void gOrYN(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(1, 8);
    OrYNBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                  in1.lwe_sample_device_, mu, fix, st.st());
}

void OrNY(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(1, 8);
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    OrNYBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                  in1.lwe_sample_device_, mu, fix, st.st());
    CtxtCopyD2H(out, st);
}

void gOrNY(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(1, 8);
    OrNYBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                  in1.lwe_sample_device_, mu, fix, st.st());
}

void And(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 8);
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    AndBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                 in1.lwe_sample_device_, mu, fix, st.st());
    CtxtCopyD2H(out, st);
}

void gAnd(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 8);
    AndBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                 in1.lwe_sample_device_, mu, fix, st.st());
}

void AndYN(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 8);
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    AndYNBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                   in1.lwe_sample_device_, mu, fix, st.st());
    CtxtCopyD2H(out, st);
}

void gAndYN(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 8);
    AndYNBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                   in1.lwe_sample_device_, mu, fix, st.st());
}

void AndNY(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 8);
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    AndNYBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                   in1.lwe_sample_device_, mu, fix, st.st());
    CtxtCopyD2H(out, st);
}

void gAndNY(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 8);
    AndNYBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                   in1.lwe_sample_device_, mu, fix, st.st());
}

void Nor(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 8);
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    NorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                 in1.lwe_sample_device_, mu, fix, st.st());
    CtxtCopyD2H(out, st);
}

void gNor(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 8);
    NorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                 in1.lwe_sample_device_, mu, fix, st.st());
}

void Xor(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(1, 4);
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    XorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                 in1.lwe_sample_device_, mu, fix, st.st());
    CtxtCopyD2H(out, st);
}

void gXor(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(1, 4);
    XorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                 in1.lwe_sample_device_, mu, fix, st.st());
}

void Xnor(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 4);
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    XnorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                  in1.lwe_sample_device_, mu, fix, st.st());
    CtxtCopyD2H(out, st);
}

void gXnor(Ctxt& out, const Ctxt& in0, const Ctxt& in1, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 4);
    XnorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
                  in1.lwe_sample_device_, mu, fix, st.st());
}

void Not(Ctxt& out, const Ctxt& in, Stream st)
{
    CtxtCopyH2D(in, st);
    NotBootstrap(out.lwe_sample_device_, in.lwe_sample_device_,
                 in.lwe_sample_->n(), st.st());
    CtxtCopyD2H(out, st);
}

void gNot(Ctxt& out, const Ctxt& in, Stream st)
{
    NotBootstrap(out.lwe_sample_device_, in.lwe_sample_device_,
                 in.lwe_sample_->n(), st.st());
}

void Copy(Ctxt& out, const Ctxt& in, Stream st)
{
    for (int i = 0; i <= in.lwe_sample_->n(); i++)
        out.lwe_sample_->data()[i] = in.lwe_sample_->data()[i];
}

void gCopy(Ctxt& out, const Ctxt& in, Stream st)
{
    hipMemcpyAsync(out.lwe_sample_device_->data(),
                    in.lwe_sample_device_->data(), in.lwe_sample_->n(),
                    hipMemcpyDeviceToDevice, st.st());
}

// Mux(inc,in1,in0) = inc?in1:in0 = inc&in1 + (!inc)&in0
void Mux(Ctxt& out, const Ctxt& ins, const Ctxt& in1, const Ctxt& in0,
         Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 8);
    static const Torus muxfix = ModSwitchToTorus(1, 8);
    CtxtCopyH2D(ins, st);
    CtxtCopyH2D(in1, st);
    CtxtCopyH2D(in0, st);
    MuxBootstrap(out.lwe_sample_device_, ins.lwe_sample_device_,
                 in1.lwe_sample_device_, in0.lwe_sample_device_, mu, fix,
                 muxfix, st.st());
    CtxtCopyD2H(out, st);
}

void gMux(Ctxt& out, const Ctxt& ins, const Ctxt& in1, const Ctxt& in0,
          Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    static const Torus fix = ModSwitchToTorus(-1, 8);
    static const Torus muxfix = ModSwitchToTorus(1, 8);
    MuxBootstrap(out.lwe_sample_device_, ins.lwe_sample_device_,
                 in1.lwe_sample_device_, in0.lwe_sample_device_, mu, fix,
                 muxfix, st.st());
}

void ConstantZero(Ctxt& out, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    for (int i = 0; i < out.lwe_sample_->n(); i++) {
        out.lwe_sample_->data()[i] = 0;
    }
    out.lwe_sample_->data()[out.lwe_sample_->n()] = -mu;
}

void gConstantZero(Ctxt& out, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    NoiselessTrivial(out.lwe_sample_device_, 0, mu, st.st());
}

void ConstantOne(Ctxt& out, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    for (int i = 0; i < out.lwe_sample_->n(); i++) {
        out.lwe_sample_->data()[i] = 0;
    }
    out.lwe_sample_->data()[out.lwe_sample_->n()] = mu;
}

void gConstantOne(Ctxt& out, Stream st)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    NoiselessTrivial(out.lwe_sample_device_, 1, mu, st.st());
}

void SetToGPU(const Ctxt& in)
{
    hipMemcpy(in.lwe_sample_device_->data(), in.lwe_sample_->data(),
               in.lwe_sample_->SizeData(), hipMemcpyHostToDevice);
}

void GetFromGPU(Ctxt& out)
{
    hipMemcpy(out.lwe_sample_->data(), out.lwe_sample_device_->data(),
               out.lwe_sample_->SizeData(), hipMemcpyDeviceToHost);
}

}  // namespace cufhe
