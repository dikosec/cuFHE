/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <include/cufhe.h>
#include <include/details/allocator_gpu.cuh>
#include <include/cufhe_gpu.cuh>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

namespace cufhe {

uint32_t cnt = 0;

Ctxt::Ctxt()
{
    std::pair<void*, MemoryDeleter> pair;
    Param* param = GetDefaultParam();

    lwe_sample_ = new LWESample(param->lwe_n_);
    lwe_sample_device_ = new LWESample(param->lwe_n_);

    pair = AllocatorCPU::New(lwe_sample_->SizeMalloc());
    lwe_sample_->set_data((LWESample::PointerType)pair.first);
    lwe_sample_deleter_ = pair.second;

/*
    pair = AllocatorGPU::New(lwe_sample_device_->SizeMalloc());
    lwe_sample_device_->set_data((LWESample::PointerType)pair.first);
    lwe_sample_device_deleter_ = pair.second;
*/

    for(int i=0;i<_gpuNum;i++){
        lwe_sample_devices_.push_back(new LWESample(param->lwe_n_));
    }

    for(int i=0;i<_gpuNum;i++){
        hipSetDevice(i);
        pair = AllocatorGPU::New(lwe_sample_devices_[i]->SizeMalloc());
        lwe_sample_devices_[i]->set_data((LWESample::PointerType)pair.first);
        lwe_sample_devices_deleter_.push_back(pair.second);
    }
}

Ctxt::~Ctxt()
{
    if (lwe_sample_ != nullptr) {
        if (lwe_sample_deleter_ != nullptr) {
            lwe_sample_deleter_(lwe_sample_->data());
            lwe_sample_deleter_ = nullptr;
        }

        lwe_sample_->set_data(nullptr);
        delete lwe_sample_;
        lwe_sample_ = nullptr;
    }

    if (lwe_sample_device_ != nullptr && lwe_sample_devices_.size() == 0) {
        if (lwe_sample_device_deleter_ != nullptr) {
            lwe_sample_device_deleter_(lwe_sample_device_->data());
            lwe_sample_device_deleter_ = nullptr;
        }

        lwe_sample_device_->set_data(nullptr);
        delete lwe_sample_device_;
        lwe_sample_device_ = nullptr;
    }

    for(int i=0;i<lwe_sample_devices_.size();i++){
        hipSetDevice(i);
        lwe_sample_devices_deleter_[i](lwe_sample_devices_[i]->data());
        lwe_sample_devices_deleter_[i] = nullptr;
    }
    lwe_sample_devices_.clear();
    lwe_sample_devices_deleter_.clear();
}

cuFHETRLWElvl1::cuFHETRLWElvl1(){
    hipHostMalloc((void**)&trlwehost,2*cuFHE_DEF_N*sizeof(Torus));
    trlwedevices.resize(_gpuNum);
    for(int i=0;i<_gpuNum;i++){
        hipSetDevice(i);
        hipMalloc((void**)&trlwedevices[i],2*cuFHE_DEF_N*sizeof(Torus));
    }
}

}  // namespace cufhe
